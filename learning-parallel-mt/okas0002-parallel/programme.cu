#include "hip/hip_runtime.h"
#include "sha512.hpp"

using SHA512_constants::b64;
using SHA512_constants::b8;


int main()
{
    validate_runtime();
    auto rand = jbutil::randgen(RANDOM_SEED);
    constexpr size_t message_length_bytes = 111 + SHA512_BLOCK_SIZE * (INPUT_SIZE_BLOCKS - 1);

    b8 * message = new b8[message_length_bytes];
    for(size_t i = 0; i < message_length_bytes / 8; ++i)
        ((b64 *) message)[i] = rand.ival64();

    std::cout << "Running at concurrency factor " << CONCURRENCY
        << " [" << OVERCOMMIT_RATIO << "x overcommited]"
        << " (" << SM_COUNT << "/" << BLOCKS_PER_SM
        << "/" << THREADS_PER_BLOCK
        << ") [multiprocessors / blocks / threads]"
        << std::endl
        ;

    // first hipMalloc is insanely slow - no point in timing that
    b64 * dhash;
    hipMalloc((void **) &dhash, 8 * sizeof(b64));

    unsigned int * dresult;
    // this could be squashed into a single hipMalloc
    // but that would only complicate the code for neglegible performance gain
    hipMalloc((void **) &dresult, sizeof(unsigned int));
    hipMemset(dresult, 0, sizeof(unsigned int));

    // ---------- clock starts here --------
    double t = jbutil::gettime();

    int padding_size = (((896 - message_length_bytes * 8 - 1) % 1024 + 1024) % 1024 + 1) / 8;

    size_t payload_size = message_length_bytes + padding_size + 16;
    b8 * payload;
    hipHostMalloc((void **) &payload, payload_size * sizeof(b8));
    // not initialized by design, since it will be overwritten anyway

    memcpy(payload, message, message_length_bytes);
    payload[message_length_bytes] = 0b10000000;
    memset(payload + message_length_bytes + 1, 0, 15);

    unsigned int size = htonl(message_length_bytes * 8);
    memcpy(payload + (payload_size - sizeof(unsigned int)), &size, sizeof(unsigned int));

    unsigned int nonce = 0;
    unsigned int result;


    int cycles = 0;
    load_payload_to_const(payload, payload_size);

    while(true)
    {
        cuda_sha512(payload, payload_size, dhash, nonce, dresult);
        hipMemcpy(&result, dresult, sizeof(unsigned int), hipMemcpyDeviceToHost);

        if (result != 0)
        {
            break;
        }

        nonce += CONCURRENCY;
        ++cycles;
    }

    b64* hash;
    hipHostMalloc((void **) &hash, 8 * sizeof(b64));
    unsigned int target_nonce = nonce + result;
    hipMemcpy(hash, dhash, 8 * sizeof(b64), hipMemcpyDeviceToHost);

    t = jbutil::gettime() - t;
    // ---------- clock stops here --------
    std::cout << "Time taken: " << t << "s" << std::endl
              << "Block size: " << message_length_bytes << " bytes" << std::endl
              << "Input block count: " << INPUT_SIZE_BLOCKS << std::endl
              << "Nonce: " << std::dec << target_nonce << std::endl
              << "Cycles: " << std::dec << cycles << std::endl
              << "MH/s " << std::dec << (nonce + CONCURRENCY) / t / 1e+6 << std::endl
              << "MHB[locks]/s " << std::dec << (nonce + CONCURRENCY) * INPUT_SIZE_BLOCKS / t / 1e+6 << std::endl
    ;

    dump_digest(hash);

    std::cout << "payload:" << std::endl;

    for (int i = 0; i < 4; ++i)
        message[i] = ((b8 *)&target_nonce)[i];

    dump_payload(message, message_length_bytes);

    // not really needed (since the OS will collect the memory anyway)
    // but here for the sake of completness
    hipFree(dhash);
    hipFree(dresult);
    delete[] message;
    hipHostFree(payload);
    hipHostFree(hash);
}


void dump_digest(b64 * hash)
{
    for(int i = 0; i < 8; ++i) 
    {
        std::cout << std::hex << std::setw(16) << std::setfill('0') << hash[i];
    }
    std::cout << std::endl;
}

void dump_payload(b8 * payload, const size_t payload_size)
{
    for(size_t i = 0; i < payload_size; ++i)
    {
        std::cout << std::hex << std::setw(2) << std::setfill('0') << (int)payload[i];
    }
    std::cout << std::endl;
}

void validate_runtime()
{
    // Little Endian check
    assertalways(htons(47) != 47);
    assertalways(sizeof(unsigned int) == 4);
}
