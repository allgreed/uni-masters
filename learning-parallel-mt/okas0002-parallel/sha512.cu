#include "hip/hip_runtime.h"
#include "sha512.hpp"

using SHA512_constants::b64;
using SHA512_constants::b8;


__global__ void ComputeHashAndCheckResult(b64 * mem_hash, unsigned int * result, const int starting_nonce, const size_t payload_size)
{
    b64 w[80];
    b64 hash[8];

    for(int i = 0; i < 8; ++i)
    {
        hash[i] = SHA512_constants::IV[i];
    }

    // unroll part of the loop and apply nonce
    b8 init[8];
    for(int i = 4; i < 8; ++i)
    {
        init[i] = dpayload[i];
    }
    unsigned int * nonce = (unsigned int *)(init);
    *nonce = starting_nonce + my_thread_id();
    for(int j = 0; j < 8; ++j)
    {
        ( (b8*)get_local_ptr_item(w, 0) )[j] = init[7 - j];
    }

    // convert to LE
    for(int i = 1; i < 16; ++i)
    {
        for(int j = 0; j < 8; ++j)
        {
            ( (b8*)get_local_ptr_item(w, i) )[j] = dpayload[7 - j + (8 * i)];
        }
    } 

    compute_round(w, hash);

    for(size_t block_offset = SHA512_BLOCK_SIZE;  block_offset < payload_size; block_offset += SHA512_BLOCK_SIZE)
    {
        for(int i = 0; i < 16; ++i)
        {
            for(int j = 0; j < 8; ++j)
            {
                ((b8 *) get_local_ptr_item(w, i))[j] = dpayload[block_offset + 7 - j + (8 * i)];
            }
        } 

        compute_round(w, hash);
    }

    //if((hash[0] > 0xcafdffffffffffff && hash[0] < 0xcaff000000000000))
    //if((hash[0] > 0xcafeafffffffffff && hash[0] < 0xcafec00000000000))
    // looking for hash starting with "cafeba"
    if (hash[0] > 0xcafeb9ffffffffff && hash[0] < 0xcafebb0000000000)
    {
        unsigned int old_result = atomicCAS(result, 0, my_thread_id());

        if (old_result == 0)
        {
            for (int i = 0; i < 8; ++i)
                mem_hash[i] = hash[i];
        }

        return;
    }
}


__device__ inline static void compute_round(b64 * w, b64 * hash)
{
    for(int i = 16; i < 80; ++i)
    {
        b64 sigma0 = 
            (d_rrot<1>(*get_local_ptr_item(w, i-15))) ^ 
            (d_rrot<8>(*get_local_ptr_item(w, i-15))) ^
            (*get_local_ptr_item(w, i-15) >> 7);

        b64 sigma1 = 
            (d_rrot<19>(*get_local_ptr_item(w, i-2))) ^
            (d_rrot<61>(*get_local_ptr_item(w, i-2))) ^
            (*get_local_ptr_item(w, i-2) >> 6);

        *get_local_ptr_item(w, i) = *get_local_ptr_item(w, i-16) + sigma0 + *get_local_ptr_item(w, i-7) + sigma1;
    }

    b64 a = *get_local_ptr_item(hash, 0);
    b64 e = *get_local_ptr_item(hash, 4);
    b64 f = *get_local_ptr_item(hash, 5);
    b64 g = *get_local_ptr_item(hash, 6);
    b64 c = *get_local_ptr_item(hash, 2);
    b64 b = *get_local_ptr_item(hash, 1);
    b64 h = *get_local_ptr_item(hash, 7);
    b64 d = *get_local_ptr_item(hash, 3);

    for(int i = 0; i < 80; ++i)
    {
        b64 Sigma0 = (d_rrot<28>(a)) ^ (d_rrot<34>(a)) ^ (d_rrot<39>(a));
        b64 Sigma1 = (d_rrot<14>(e)) ^ (d_rrot<18>(e)) ^ (d_rrot<41>(e));
        b64 ch = (e & f) ^ ((~ e) & g);
        b64 maj = (a & b) ^ (a & c) ^ (b & c);

        b64 temp1 = h + Sigma1 + ch + SHA512_constants::K[i] + *get_local_ptr_item(w, i);
        b64 temp2 = Sigma0 + maj;

        h = g;
        g = f;
        f = e;
        e = d + temp1;
        d = c;
        c = b;
        b = a;
        a = temp1 + temp2;
    }

    *get_local_ptr_item(hash, 0) += a;
    *get_local_ptr_item(hash, 1) += b;
    *get_local_ptr_item(hash, 2) += c;
    *get_local_ptr_item(hash, 3) += d;
    *get_local_ptr_item(hash, 4) += e;
    *get_local_ptr_item(hash, 5) += f;
    *get_local_ptr_item(hash, 6) += g;
    *get_local_ptr_item(hash, 7) += h;
}


void load_payload_to_const(b8 * payload, const size_t payload_size)
{
    // for reasons foreing and bizzare to me
    // this memcpy has to happen in this file
    // belive me, I tried
    hipMemcpyToSymbol(HIP_SYMBOL(dpayload), payload, payload_size);
}


void cuda_sha512(b8 * payload, const size_t payload_size, b64 * dhash, const int starting_nonce, unsigned int * dresult)
{
    ComputeHashAndCheckResult<<<BLOCK_COUNT, THREADS_PER_BLOCK>>>(dhash, dresult, starting_nonce, payload_size);
}


template <unsigned char D>
__device__ inline static b64 d_rrot(b64 n)
{ 
    static_assert(D <= sizeof(b64) * 8, "unsupported value for rotation parameter D, D is too large");

    return (n >> D) | (n << (sizeof(b64) * 8 - D));
}


__device__ inline static unsigned int my_thread_id()
{
    return blockIdx.x*blockDim.x + threadIdx.x;
}

template <typename T>
__device__ inline static T get_local_ptr_item(T ptr, size_t idx)
{
    return get_remote_ptr_item(ptr, idx, my_thread_id());
}

template <typename T>
__device__ T get_remote_ptr_item(T ptr, size_t idx, unsigned int unique_thread_id)
{
    return ptr + idx;
}
